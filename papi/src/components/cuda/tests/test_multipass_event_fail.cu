/**
 * @file    test_multipass_event_fail.cu
 * @author  Anustuv Pal
 *          anustuv@icl.utk.edu
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#ifdef PAPI
#include "papi.h"
#include "papi_test.h"

#define PASS 1
#define FAIL 0
#define MAX_EVENT_COUNT (32)
#define PRINT(quiet, format, args...) {if (!quiet) {fprintf(stderr, format, ## args);}}
int quiet;

int test_PAPI_add_named_event(int *EventSet, int numEvents, char **EventName) {
    int i, papi_errno;
    PRINT(quiet, "LOG: %s: Entering.\n", __func__);
    for (i=0; i<numEvents; i++) {
        papi_errno = PAPI_add_named_event(*EventSet, EventName[i]);
        if (papi_errno == PAPI_ENOEVNT) {
            fprintf(stderr, "Event name %s does not exist.\n", EventName[i]);
            return FAIL;
        }
        if (papi_errno != PAPI_OK) {
            PRINT(quiet, "Error %d: Failed to add event %s\n", papi_errno, EventName[i]);
        }
    }
    if (papi_errno == PAPI_EMULPASS)
        return PASS;           // Test pass condition
    return FAIL;
}

int test_PAPI_add_event(int *EventSet, int numEvents, char **EventName) {
    int event, i, papi_errno;
    PRINT(quiet, "LOG: %s: Entering.\n", __func__);

    for (i=0; i<numEvents; i++) {
        papi_errno = PAPI_event_name_to_code(EventName[i], &event);
        if (papi_errno == PAPI_ENOEVNT) {
            fprintf(stderr, "Event name %s does not exist.\n", EventName[i]);
            return FAIL;
        }
        if (papi_errno != PAPI_OK) {
            PRINT(quiet, "Error %d: Error in name to code.\n", papi_errno);
            goto fail;
        }
        papi_errno = PAPI_add_event(*EventSet, event);
        if (papi_errno != PAPI_OK) {
            PRINT(quiet, "Error %d: Failed to add event %s\n", papi_errno, EventName[i]);
        }
    }
    if (papi_errno == PAPI_EMULPASS)
        return PASS;
fail:
    return FAIL;
}

int test_PAPI_add_events(int *EventSet, int numEvents, char **EventName) {
    int papi_errno, i;
    PRINT(quiet, "LOG: %s: Entering.\n", __func__);

    int events[MAX_EVENT_COUNT];

    for (i=0; i<numEvents; i++) {
        papi_errno = PAPI_event_name_to_code(EventName[i], &events[i]);
        if (papi_errno != PAPI_OK) {
            PRINT(quiet, "Error %d: Error in name to code.\n", papi_errno);
            goto fail;
        }
    }
    papi_errno = PAPI_add_events(*EventSet, events, numEvents);
    if (papi_errno != PAPI_OK) {
        PRINT(quiet, "Error %d: Failed to add %d events\n", papi_errno, numEvents);
    }
    if (papi_errno < numEvents)        // Returns index at which error occurred.
        return PASS;
fail:
    return FAIL;
}
#endif

int main(int argc, char **argv)
{
#ifdef PAPI
    int papi_errno, pass;
    int event_set;

    quiet = 0;
    char *test_quiet = getenv("PAPI_CUDA_TEST_QUIET");
    if (test_quiet)
        quiet = (int) strtol(test_quiet, (char**) NULL, 10);

    int event_count = argc - 1;

    /* if no events passed at command line, just report test skipped. */
    if (event_count == 0) {
        fprintf(stderr, "No eventnames specified at command line.\n");
        test_skip(__FILE__, __LINE__, "", 0);
    }

    papi_errno = PAPI_library_init( PAPI_VER_CURRENT );
    if (papi_errno != PAPI_VER_CURRENT) {
        test_fail(__FILE__, __LINE__, "PAPI_library_init() failed", 0);
    }

    papi_errno = PAPI_get_component_index("cuda");
    if (papi_errno < 0 ) {
        test_fail(__FILE__, __LINE__, "CUDA component not configured", 0);
    }

    event_set = PAPI_NULL;
    papi_errno = PAPI_create_eventset( &event_set );
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_create_eventset() failed!", 0);
    }

    pass = test_PAPI_add_event(&event_set, argc-1, argv+1);
    papi_errno = PAPI_cleanup_eventset(event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset() failed!", 0);
    }

    papi_errno = PAPI_destroy_eventset(&event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset() failed!", 0);
    }

    event_set = PAPI_NULL;
    papi_errno = PAPI_create_eventset( &event_set );
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_create_eventset() failed!", 0);
    }

    pass += test_PAPI_add_named_event(&event_set, argc-1, argv+1);
    papi_errno = PAPI_cleanup_eventset(event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset() failed!", 0);
    }

    papi_errno = PAPI_destroy_eventset(&event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset() failed!", 0);
    }

    event_set = PAPI_NULL;
    papi_errno = PAPI_create_eventset( &event_set );
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_create_eventset() failed!", 0);
    }

    pass += test_PAPI_add_events(&event_set, argc-1, argv+1);
    papi_errno = PAPI_cleanup_eventset(event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset() failed!", 0);
    }

    papi_errno = PAPI_destroy_eventset(&event_set);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset() failed!", 0);
    }

    if (pass != 3)
        test_fail(__FILE__, __LINE__, "CUDA framework multipass event test failed.", 0);
    else
        test_pass(__FILE__);

    PAPI_shutdown();
#else
    fprintf(stderr, "Please compile with -DPAPI to test this feature.\n");
#endif
    return 0;
}