#include "hip/hip_runtime.h"
/****************************/
/* THIS IS OPEN SOURCE CODE */
/****************************/

/** 
 * @file    HelloWorld.c
 * CVS:     $Id$
 * @author  Heike Jagode
 *          jagode@eecs.utk.edu
 * Mods:	<your name here>
 *			<your email address>
 * test case for Example component 
 * 
 *
 * @brief
 *  This file is a very simple HelloWorld C example which serves (together
 *	with its Makefile) as a guideline on how to add tests to components.
 *  The papi configure and papi Makefile will take care of the compilation
 *	of the component tests (if all tests are added to a directory named
 *	'tests' in the specific component dir).
 *	See components/README for more details.
 *
 *	The string "Hello World!" is mangled and then restored.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "papi.h"
#include "papi_test.h"

#define NUM_EVENTS 1
#define PAPI

// Prototypes
__global__ void helloWorld(char*);


// Host function
int main(int argc, char** argv)
{
#ifdef PAPI
	int retval, i;
	int EventSet = PAPI_NULL;
	long long values[NUM_EVENTS];
	/* REPLACE THE EVENT NAME 'PAPI_FP_OPS' WITH A CUDA EVENT 
	   FOR THE CUDA DEVICE YOU ARE RUNNING ON.
	   RUN papi_native_avail to get a list of CUDA events that are 
	   supported on your machine */
        // e.g. on a P100 nvml:::Tesla_P100-SXM2-16GB:power
        char anEvent[64] = "PAPI_FP_OPS";
        char *EventName[] = { anEvent };
        int events[NUM_EVENTS];
	int eventCount = 0;
	
	/* PAPI Initialization */
	retval = PAPI_library_init( PAPI_VER_CURRENT );
	if( retval != PAPI_VER_CURRENT )
		fprintf( stderr, "PAPI_library_init failed\n" );
	
	printf( "PAPI_VERSION     : %4d %6d %7d\n",
			PAPI_VERSION_MAJOR( PAPI_VERSION ),
			PAPI_VERSION_MINOR( PAPI_VERSION ),
			PAPI_VERSION_REVISION( PAPI_VERSION ) );
	
	/* convert PAPI native events to PAPI code */
	for( i = 0; i < NUM_EVENTS; i++ ){
		retval = PAPI_event_name_to_code( EventName[i], &events[i] );
		if( retval != PAPI_OK ) {
			fprintf( stderr, "PAPI_event_name_to_code failed\n" );
			continue;
		}
		eventCount++;
			printf( "Name %s --- Code: %#x\n", EventName[i], events[i] );
	}

	/* if we did not find any valid events, just report test failed. */
	if (eventCount == 0) {
		printf( "Test FAILED: no valid events found.\n");
		return 1;
	}
	
	retval = PAPI_create_eventset( &EventSet );
	if( retval != PAPI_OK )
		fprintf( stderr, "PAPI_create_eventset failed\n" );
	
	retval = PAPI_add_events( EventSet, events, eventCount );
	if( retval != PAPI_OK )
		fprintf( stderr, "PAPI_add_events failed\n" );
#endif


	int j;
	int count;
	int cuda_device;

	hipGetDeviceCount( &count );
	for ( cuda_device = 0; cuda_device < count; cuda_device++ ) {
			hipSetDevice( cuda_device );
#ifdef PAPI	
			retval = PAPI_start( EventSet );
			if( retval != PAPI_OK )
					fprintf( stderr, "PAPI_start failed\n" );
#endif

			// desired output
			char str[] = "Hello World!";

			// mangle contents of output
			// the null character is left intact for simplicity
			for(j = 0; j < 12; j++) {
					str[j] -= j;
					//printf("str=%s\n", str);
			}


			// allocate memory on the device
			char *d_str;
			size_t size = sizeof(str);
			hipMalloc((void**)&d_str, size);

			// copy the string to the device
			hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

			// set the grid and block sizes
			dim3 dimGrid(2); // one block per word
			dim3 dimBlock(6); // one thread per character


			// invoke the kernel
			helloWorld<<< dimGrid, dimBlock >>>(d_str);

			// retrieve the results from the device
			hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

			// free up the allocated memory on the device
			hipFree(d_str);

			printf("END: %s\n", str);


#ifdef PAPI
			retval = PAPI_stop( EventSet, values );
			if( retval != PAPI_OK )
					fprintf( stderr, "PAPI_stop failed\n" );

			for( i = 0; i < eventCount; i++ )
					printf( "On device %d: %12lld \t\t --> %s \n", cuda_device, values[i], EventName[i] );
#endif
	}

	return 0;
}


// Device kernel
__global__ void
helloWorld(char* str)
{
	// determine where in the thread grid we are
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// unmangle output
	str[idx] += idx;
}

